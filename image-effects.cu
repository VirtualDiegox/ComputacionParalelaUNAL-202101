#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/core.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <iostream>
#include <stdlib.h>
#include <sys/time.h>
#include <stdio.h>
#include <string>

using namespace std;
using namespace cv;

__global__ void grisPromedioCUDA(uint8_t * Ptr_src,uint8_t * Ptr_dst, int *bloques, int *height, int *width,int step){
    int index = threadIdx.x + blockIdx.x * blockDim.x; 
    int hilos = blockDim.x * (*bloques); 
    uint8_t pixel[3];
    int pixelsPerThread = (*height)*(*width) / hilos;
    //segun la cantidad de hilos dividimos las iteraciones por block-wise y se las pasamos a una estructura para pasar la info al hilo
    int inicioy = (int)((pixelsPerThread * index)/(*width) ) ;
    int finy = (int)((pixelsPerThread * (index + 1))/(*width));

    int iniciox = (int)((pixelsPerThread * index)%(*width));
    int finx = (int)((pixelsPerThread * (index+1))%(*width));
    
    //if (index == (hilos - 1)){
       // fin = (*height) - 1;
    //}
    int ancho = *width;
    int paso = step;
    
    for (int i = inicioy; i <= finy; i++){
        for (int j = ( i == inicioy ) ? iniciox : 0 ; j < ((i == finy)? finx : ancho); j++){
            //obtenemos valores RGB de la imagen
            pixel[0] = (uint8_t)(Ptr_src[i * paso * 3  + j * 3 + 0]); // B
            pixel[1] = (uint8_t)(Ptr_src[i * paso * 3 + j * 3 + 1]); // G
            pixel[2] = (uint8_t)(Ptr_src[i * paso * 3 + j * 3 + 2]); // R
            

            //hacemos la logica del filtro con los valores RGB
            uint8_t Grey = (pixel[0] + pixel[1] + pixel[2]) / 3;
            //asignamos el valor calculado al unico canal de la imagen a crear
           
            Ptr_dst[i * paso + j] = Grey;
        };
     
    };
    
}
__global__ void grisLumaCUDA(uint8_t * Ptr_src,uint8_t * Ptr_dst, int *bloques, int *height, int *width,int step){
    int index = threadIdx.x + blockIdx.x * blockDim.x; 
    int hilos = blockDim.x * (*bloques); 
    uint8_t pixel[3];
    int pixelsPerThread = (*height)*(*width) / hilos;
    //segun la cantidad de hilos dividimos las iteraciones por block-wise y se las pasamos a una estructura para pasar la info al hilo
    int inicioy = (int)((pixelsPerThread * index)/(*width) ) ;
    int finy = (int)((pixelsPerThread * (index + 1))/(*width));

    int iniciox = (int)((pixelsPerThread * index)%(*width));
    int finx = (int)((pixelsPerThread * (index+1))%(*width));
    
    
    //if (index == (hilos - 1)){
        //fin = (*height) - 1;
    //}
    int ancho = *width;
    int paso = step;
    
    for (int i = inicioy; i <= finy; i++){
        for (int j = ( i == inicioy ) ? iniciox : 0 ; j < ((i == finy)? finx : ancho); j++){
            //obtenemos valores RGB de la imagen
            pixel[0] = (uint8_t)(Ptr_src[i * paso * 3  + j * 3 + 0]); // B
            pixel[1] = (uint8_t)(Ptr_src[i * paso * 3 + j * 3 + 1]); // G
            pixel[2] = (uint8_t)(Ptr_src[i * paso * 3 + j * 3 + 2]); // R
            

            //hacemos la logica del filtro con los valores RGB
            uint8_t Grey = (pixel[0]*0.0722+pixel[1]*0.7152+pixel[2]*0.2126);
            //asignamos el valor calculado al unico canal de la imagen a crear
           
            Ptr_dst[i * paso + j] = Grey;
        };
     
    };
    
}

__global__ void sombrasDeGrisCUDA(uint8_t * Ptr_src,uint8_t * Ptr_dst, int *bloques, int *height, int *width,int step,int *capas){
    int index = threadIdx.x + blockIdx.x * blockDim.x; 
    int hilos = blockDim.x * (*bloques); 
    uint8_t pixel[3];
    if(*capas<2) *capas = 2;
    if(*capas>255) *capas = 255;
    int ConversionFactor = 255 / (*capas - 1);
    //segun la cantidad de hilos dividimos las iteraciones por block-wise y se las pasamos a una estructura para pasar la info al hilo
    int pixelsPerThread = (*height)*(*width) / hilos;
    
    int inicioy = (int)((pixelsPerThread * index)/(*width) ) ;
    int finy = (int)((pixelsPerThread * (index + 1))/(*width));

    int iniciox = (int)((pixelsPerThread * index)%(*width));
    int finx = (int)((pixelsPerThread * (index+1))%(*width));

    //if (index == (hilos - 1)){
        //finy = (*height) - 1;
    //}
    
    int ancho = *width;
    int paso = step;
    
    for (int i = inicioy; i <= finy; i++){

        for (int j = ( i == inicioy ) ? iniciox : 0 ; j < ((i == finy)? finx : ancho); j++){


            //obtenemos valores RGB de la imagen
            pixel[0] = (uint8_t)(Ptr_src[i * paso * 3  + j * 3 + 0]); // B
            pixel[1] = (uint8_t)(Ptr_src[i * paso * 3 + j * 3 + 1]); // G
            pixel[2] = (uint8_t)(Ptr_src[i * paso * 3 + j * 3 + 2]); // R
            

            //hacemos la logica del filtro con los valores RGB
            uint8_t AverageValue = (pixel[0]+pixel[1]+pixel[2])/3;
            uint8_t Grey = (int)((AverageValue / ConversionFactor) + 0.5) * ConversionFactor;
            //asignamos el valor calculado al unico canal de la imagen a crear
           
            Ptr_dst[i * paso + j] = Grey;
        };
    };  
}




void grisPromedio(Mat Ptr_src, Mat Ptr_dst, int hilos, int bloques)
{

    struct timeval *tval_before, *tval_after, *tval_result;
    tval_before = (struct timeval *)malloc(sizeof(struct timeval));
    tval_after = (struct timeval *)malloc(sizeof(struct timeval));
    tval_result = (struct timeval *)malloc(sizeof(struct timeval));

    int cn = Ptr_src.channels();//Numero de canales    
   
    cv::cuda::GpuMat d_src{ Ptr_src.rows, Ptr_src.cols, 1000};
    cv::cuda::GpuMat d_dst{ Ptr_src.rows, Ptr_src.cols, 1000};
    cv::cuda::createContinuous(Ptr_src.rows, Ptr_src.cols,CV_8UC3,d_src);
    cv::cuda::createContinuous(Ptr_src.rows, Ptr_src.cols,CV_8UC1,d_dst);
    
    d_src.upload(Ptr_src);
    d_dst.upload(Ptr_dst);


    int height = Ptr_src.rows;
    int width = Ptr_dst.cols;
    int step = (int)Ptr_src.step/(int)sizeof(uint8_t);
    int size = Ptr_src.rows * Ptr_src.step;
    

    //CUDA
   
    int *d_height, *d_bloques, *d_width,*d_step;

    
    hipMalloc((void **)&d_height, sizeof(int));
    hipMalloc((void **)&d_width, sizeof(int));
    hipMalloc((void **)&d_bloques, sizeof(int));
    hipMalloc((void **)&d_step, sizeof(int));


    hipMemcpy(d_height, &height, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_width, &width, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_bloques, &bloques, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_step, &step, sizeof(int), hipMemcpyHostToDevice);
    hipStream_t stream;
    hipStreamCreate(&stream);
    gettimeofday(tval_before, NULL);
    grisPromedioCUDA<<<bloques, hilos,0,stream>>>((uint8_t*)d_src.data, (uint8_t*)d_dst.data, d_bloques, d_height, d_width,d_dst.step);
    hipStreamSynchronize(stream);
    gettimeofday(tval_after, NULL);
    timersub(tval_after, tval_before, tval_result);
    printf("%ld.%06ld\n", (long int)tval_result->tv_sec, (long int)tval_result->tv_usec);
    
    d_dst.download(Ptr_dst);
    hipStreamDestroy(stream);
    hipFree(d_bloques);
    hipFree(d_height);
    hipFree(d_width);
    hipFree(d_step);
}

void grisLuma(Mat Ptr_src, Mat Ptr_dst, int hilos, int bloques)
{

    struct timeval *tval_before, *tval_after, *tval_result;
    tval_before = (struct timeval *)malloc(sizeof(struct timeval));
    tval_after = (struct timeval *)malloc(sizeof(struct timeval));
    tval_result = (struct timeval *)malloc(sizeof(struct timeval));
    int cn = Ptr_src.channels();//Numero de canales    
    
    cv::cuda::GpuMat d_src{ Ptr_src.rows, Ptr_src.cols, 1000};
    cv::cuda::GpuMat d_dst{ Ptr_src.rows, Ptr_src.cols, 1000};
    cv::cuda::createContinuous(Ptr_src.rows, Ptr_src.cols,CV_8UC3,d_src);
    cv::cuda::createContinuous(Ptr_src.rows, Ptr_src.cols,CV_8UC1,d_dst);
    
    d_src.upload(Ptr_src);
    d_dst.upload(Ptr_dst);

    int height = Ptr_src.rows;
    int width = Ptr_dst.cols;
    int step = (int)Ptr_src.step/(int)sizeof(uint8_t);
    int size = Ptr_src.rows * Ptr_src.step;
    

    //CUDA
   
    int *d_height, *d_bloques, *d_width,*d_step;

    
    hipMalloc((void **)&d_height, sizeof(int));
    hipMalloc((void **)&d_width, sizeof(int));
    hipMalloc((void **)&d_bloques, sizeof(int));
    hipMalloc((void **)&d_step, sizeof(int));


    hipMemcpy(d_height, &height, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_width, &width, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_bloques, &bloques, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_step, &step, sizeof(int), hipMemcpyHostToDevice);
    hipStream_t stream;
    hipStreamCreate(&stream);
    gettimeofday(tval_before, NULL);
    grisLumaCUDA<<<bloques, hilos,0,stream>>>((uint8_t*)d_src.data, (uint8_t*)d_dst.data, d_bloques, d_height, d_width,d_dst.step);
    hipStreamSynchronize(stream);
    gettimeofday(tval_after, NULL);
    timersub(tval_after, tval_before, tval_result);
    printf("%ld.%06ld\n", (long int)tval_result->tv_sec, (long int)tval_result->tv_usec);
    
    d_dst.download(Ptr_dst);
    hipStreamDestroy(stream);
    hipFree(d_bloques);
    hipFree(d_height);
    hipFree(d_width);
    hipFree(d_step);
}

void sombrasDeGris(Mat Ptr_src, Mat Ptr_dst, int hilos, int bloques,int capas)
{

    struct timeval *tval_before, *tval_after, *tval_result;
    tval_before = (struct timeval *)malloc(sizeof(struct timeval));
    tval_after = (struct timeval *)malloc(sizeof(struct timeval));
    tval_result = (struct timeval *)malloc(sizeof(struct timeval));
    int cn = Ptr_src.channels();//Numero de canales    
   
    cv::cuda::GpuMat d_src{ Ptr_src.rows, Ptr_src.cols, 1000};
    cv::cuda::GpuMat d_dst{ Ptr_src.rows, Ptr_src.cols, 1000};
    cv::cuda::createContinuous(Ptr_src.rows, Ptr_src.cols,CV_8UC3,d_src);
    cv::cuda::createContinuous(Ptr_src.rows, Ptr_src.cols,CV_8UC1,d_dst);
    
    d_src.upload(Ptr_src);
    d_dst.upload(Ptr_dst);

    int height = Ptr_src.rows;
    int width = Ptr_dst.cols;
    int step = (int)Ptr_src.step/(int)sizeof(uint8_t);
    int size = Ptr_src.rows * Ptr_src.step;
    

    //CUDA
   
    int *d_height, *d_bloques, *d_width,*d_step,*d_capas;

    
    hipMalloc((void **)&d_height, sizeof(int));
    hipMalloc((void **)&d_width, sizeof(int));
    hipMalloc((void **)&d_bloques, sizeof(int));
    hipMalloc((void **)&d_step, sizeof(int));
    hipMalloc((void **)&d_capas, sizeof(int));



    hipMemcpy(d_height, &height, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_width, &width, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_bloques, &bloques, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_step, &step, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_capas, &capas, sizeof(int), hipMemcpyHostToDevice);
    hipStream_t stream;
    hipStreamCreate(&stream);
    gettimeofday(tval_before, NULL);
    sombrasDeGrisCUDA<<<bloques, hilos,0,stream>>>((uint8_t*)d_src.data, (uint8_t*)d_dst.data, d_bloques, d_height, d_width,d_dst.step,d_capas);
    hipStreamSynchronize(stream);
    gettimeofday(tval_after, NULL);
    timersub(tval_after, tval_before, tval_result);
    printf("%ld.%06ld\n", (long int)tval_result->tv_sec, (long int)tval_result->tv_usec);
    
    d_dst.download(Ptr_dst);
    hipStreamDestroy(stream);
    hipFree(d_bloques);
    hipFree(d_height);
    hipFree(d_width);
    hipFree(d_step);
    hipFree(d_capas);
}


int main(int argc, char *argv[])
{
    
    //declaramos argumentos de entradas
    char *nombre_src;
    nombre_src = (char *)malloc(sizeof(char) * 20);
    char *nombre_dst;
    nombre_dst = (char *)malloc(sizeof(char) * 20);
    int parametro_filtro, capas, hilos, bloques;
    //tomando argumentos por consola
    strcpy(nombre_src, argv[1]);
    strcpy(nombre_dst, argv[2]);
    parametro_filtro = stoi(argv[3]);
    if (parametro_filtro == 3 || parametro_filtro == 4)
    {
        capas = stoi(argv[4]);
        hilos = stoi(argv[5]);
        bloques = stoi(argv[6]);
    }
    else
    {
        hilos = stoi(argv[4]);
        bloques = stoi(argv[5]);
    }

    //caso en el que se quiera correr el filtro secuencialmente

    //declaramos variable para tomar el tiempo
    //struct timeval *tval_before, *tval_after, *tval_result;
    //tval_before = (struct timeval *)malloc(sizeof(struct timeval));
    //tval_after = (struct timeval *)malloc(sizeof(struct timeval));
    //tval_result = (struct timeval *)malloc(sizeof(struct timeval));
    //Leemos la imagen
    Mat imagen_src = imread(nombre_src);
    free(nombre_src);
    //Advertimos si no se encuentra la imagen
    if (imagen_src.empty())
    {
        printf(" Error opening image\n");
        return -1;
    }

    //Declaramos objeto sobre el cual se trabajara la imagen a crear
    Mat image_dst(imagen_src.rows, imagen_src.cols, CV_8UC1, Scalar(0));

    int height = imagen_src.rows;
    int width = imagen_src.cols;

    switch (parametro_filtro)
    {
    //filtro grisPromedio
    case 1:
        grisPromedio(imagen_src, image_dst, hilos, bloques);
        break;
    //filtro Luma
    case 2:
        
        grisLuma(imagen_src,image_dst, hilos, bloques);
        
        break;
    //filtro sombrasDeGris
    case 3:
        
        sombrasDeGris(imagen_src,image_dst, hilos, bloques,capas);
        
        break;
    //filtro granular
    case 4:
        //gettimeofday(tval_before, NULL);
        //granular(imagen_src,image_dst,capas);
        //gettimeofday(tval_after, NULL);
        //timersub(tval_after, tval_before, tval_result);
        //printf("%ld.%06ld\n", (long int)tval_result->tv_sec, (long int)tval_result->tv_usec);
        break;
    default:
        break;
    }
    //Guardamos la imagen
    imwrite(nombre_dst, image_dst);
    free(nombre_dst);
    return 1;

    //40 Multiprocessors 128 hilos
}
