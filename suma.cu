#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <opencv2/cudaimgproc.hpp>



__global__ void add(int* a,int* b,int* c){
    *c = *a + *b;
    printf("sumando");
}


int main(){
    printf("%s",CV_VERSION);
    int a,b,c;
    int *d_a, *d_b, *d_c;
    int size = sizeof(int);
    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&d_b,size);
    hipMalloc((void**)&d_c,size);
    a = 4;
    b = 5;
    c = 0;
    hipMemcpy(d_a,&a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,&b,size,hipMemcpyHostToDevice);

    add<<<1,1>>>(d_a,d_b,d_c);

    hipMemcpy(&c,d_c,size,hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    printf("%d\n",c);

    return 0;




}